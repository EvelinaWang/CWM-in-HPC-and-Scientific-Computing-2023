//include libraries
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

//define constant
#define PI 3.1415926
#define NUM_ELS	 50

//my kernel
__global__ void mc_pi(float *d_x, float *d_y, int *res){

    //get varaibles from array  
    int index=blockIdx.x*blockDim.x + threadIdx.x;
    float x=d_x[index];
    float y=d_y[index];
   
    //count the number  of elements inside range of circle
    if(x*x + y*y <= 1.0f) {
        atomicAdd(res,1);
    }
}


//main function
int main() {
  
    //declare variables in CPU
    int *h_res=NULL;
    //float *h_x=NULL;
    //float *h_y=NULL;

    //allocate memory in CPU
    h_res = (int*) malloc(1*sizeof(int));
    //h_x = (float*) malloc(NUM_ELS*sizeof(float));
    //h_y = (float*) malloc(NUM_ELS*sizeof(float));

    //generate random variables using curand generator
    //curandGenerator_t gen;

    //Create random number generator
    //curandCreateGenerator(&gen, CURAND_RNG_PSEUDO_DEFAULT);

    //Set the generator options
    //curandSetPseudoRandomGeneratorSeed(gen,1234ULL);

    //generate the randoms
    //curandGenerateUniform(gen, h_x, NUM_ELS);
    //curandGenerateUniform(gen, h_y, NUM_ELS);

    //initialise GPU
    int deviceid=0;
    int devCount=0;

    //count the maximum id number 
    hipGetDeviceCount(&devCount);

    //error checking
    if(deviceid<devCount){
      hipSetDevice(deviceid);
     }else return 1;

    //define variable
    size_t array_size = NUM_ELS * sizeof(float);
    size_t int_size = 1 * sizeof(int);
    float *d_x;
    float *d_y;
    int *res;

    //allocate memory in GPU
    hipMalloc((void**)&d_x,array_size);
    hipMalloc((void**)&d_y,array_size);
    hipMalloc((void**)&res,int_size);
    hipMemset(res, 0, int_size);
    
    //generate random variables using curand generator
    hiprandGenerator_t gen;

    //Create random number generator
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

    //Set the generator options
    hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);

    //generate the randoms
    hiprandGenerateUniform(gen, d_x, NUM_ELS);
    hiprandGenerateUniform(gen, d_y, NUM_ELS);

    //transfer data to GPU
    //cudaMemcpy(d_y,h_y,NUM_ELS*sizeof(float),cudaMemcpyHostToDevice);
    //cudaMemcpy(d_x,h_x,NUM_ELS*sizeof(float),cudaMemcpyHostToDevice);

    //run the kernel
    mc_pi<<<10,5>>>(d_x, d_y,res); 

    //transfer data back to CPU
    hipMemcpy(h_res,res,1*sizeof(float),hipMemcpyDeviceToHost);

    //print out the answer
    printf("\nresult is:\t%f\n", 4.0*(*h_res)/NUM_ELS);
  
    //free memories
    hipFree(d_x);
    hipFree(d_y);
    hipFree(res);
    //free(h_x);
    //free(h_y);
    free(h_res);

    return(0);
}

