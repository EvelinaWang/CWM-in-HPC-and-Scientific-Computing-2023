//random number generation
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <stdio.h>

//define variables
#define NUM_ELS=50;

//local functions
float mean(float *array, int n);
float std_dev(float *array, int n, float mean);
void print_Histogram(float *array, int n);

int main(void){
    //Declare variable
    hiprandGenerator_t gen;

    //Create random number generator
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

    //Set the generator options
    hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);

    //allocate memory in GPU
    float *d_input;
    array_size = (float*) malloc(NUM_ELS*sizeof(float));
    hipMalloc((void**)&d_input,array_size);

    //generate the randoms
    hiprandGenerateNormal(gen, d_input, NUM_ELS, 0.0f,1.0f);

    //transfer data to CPU
    float *h_input;
    hipMemcpy(h_input, d_input, array_size,cudaDeviceToHost);

    //free GPU memory
    hipFree(d_input);

    for(int i=0;i<NUM_ELS;i++){
        printf("%f\n",h_input[i]);
    }

    //use local functions to get values
    float mean_value=mean(h_input,NUM_ELS);
    float standard_deviation=std_dev(h_input,NUM_ELS,mean_value);
    printf("mean is %f and standard deviation is %f\n", mean_value, standard_deviaiton);
    print_Histogram(h_input, NUM_ELS);

    return 0;
}

//function to calculate mean
float mean(float *array, int n){
    float sum=0;
    for(int i=0;i<n;i++){
        sum+=array[i];
    }
    return sum/n;
}

//function to calculate standart deviation
float std_dev(float *array, int n, float mean){
    float sum_2=0;
    for(int i=0;i<n;i++){
        sum_2+=array[i]*array[i];
    }
    std=(sum_2/n)-(mean*mean);
    return std;
}

//print out the histogram of random numbers generated
void print_Histogram(float *array, int n){
    printf("Histogram\n");
    for(int i=0;i<n;i++){
        for(int j=0; j<array[i];j++){
            printf("#");
        }
        printf("\n");
    }
}
