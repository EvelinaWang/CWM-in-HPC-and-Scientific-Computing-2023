// In this assignment you will write a basic kernel where every thread 
// will write out to console string "Hello world!".
// You will also initialize GPU using cudaSetDevice() and also launch
// your "Hello world" kernel.

#include <stdio.h>
#include <stdlib.h>
// we have to include few more things
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

//----------------------------------------------------------------------
// TASK 2: Write a "Hello world" kernel

__global__ void helloworld_GPU(void){
    printf("hello world!\n");
} 
// Remember that kernel is defined by __global__ and inside it looks like
// a serial code for CPU. For printing out to console you can use printf()

//----------------------------------------------------------------------

int main(void) {
  //----------------------------------------------------------------------
  // TASK 1: Initiate GPU using cudaSetDevice()
  //
  // You can also try to write a check if there is a device with that id,
  // so the code behaves nicely when it fails
  
    int deviceid=0;
    int devCount;

    hipGetDeviceCount(&devCount);

    if(deviceid<devCount){
        hipSetDevice(deviceid);
    }else return 1;

  //--------------------------------------------------------------------
  
  
  //----------------------------------------------------------------------
  // TASK 3: execute your "Hello world" kernel on 1 block with 5 threads 
  //         using execution configuration syntax.
  // 
  // You may use whatever syntax version you prefer, a simplified one 
  // dimensional or full three dimensional call using dim3 data type.
  
  // execute your "Hello world" kernel here
    helloworld_GPU<<<1,5>>>();  
  //----------------------------------------------------------------------
 
  hipDeviceReset(); 
  return (0);
}
